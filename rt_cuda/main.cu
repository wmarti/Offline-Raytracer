#include "hip/hip_runtime.h"
#include <iostream>
#include "vec3.cuh"
#include "ray.cuh"
#include "Timer.h"
#include "sphere.cuh"
#include "hittable.cuh"
#include "hittable_list.cuh"
#include "camera.cuh"
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "material.cuh"

#define TILE_SIZE 8
#define ASPECT_RATIO 16.0 / 9.0
#define IMAGE_WIDTH 1920
#define SAMPLES_PER_PIXEL 20
#define MAX_DEPTH 50

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__);  }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
    hipGetErrorString(code), file, line);
    if(abort)
        exit(code);
    }
}

__device__ color ray_color(const ray& r, hittable** world, int max_depth, hiprandState* local_rand_state)
{
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < max_depth; i++) {
       hit_record rec;
       if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
        ray scattered;
        vec3 attenuation;
        if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
            cur_attenuation *= attenuation;
            cur_ray = scattered;
        }
        else {
            return vec3(0.0,0.0,0.0);
        }
    }
       else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
         }
       }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hittable **d_list, hittable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0,-1000.0,-1), 1000,
                               new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a+RND,0.2,b+RND);
                if(choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                }
                else if(choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new hittable_list(d_list, 22*22+1+3);

        vec3 lookfrom(13,2,3);
        vec3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

__global__ void clean_up(hittable **d_list, hittable **d_world, camera** d_camera) {
    for(int i=0; i < 22*22+1+3; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
 }

 __global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void init_rand_state(int image_width, int image_height, hiprandState* rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= image_width) || (j >= image_height)) return;
    int pixel_index = j * image_width + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* frame_buffer, int image_width, int image_height, int samples_per_pixel,
                       camera** d_camera, hittable** d_world, hiprandState* rand_state, int max_depth)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= image_width) || (j >= image_height)) return;
    int pixel_index = j * image_width + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    color pixel_color(0, 0, 0);
    for(int s = 0; s < samples_per_pixel; s++)
    {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(image_width);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(image_height);
        ray r = (*d_camera)->get_ray(u, v, &local_rand_state);
        pixel_color += ray_color(r, d_world, max_depth, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    pixel_color /= float(samples_per_pixel);
    pixel_color[0] = sqrt(pixel_color[0]);
    pixel_color[1] = sqrt(pixel_color[1]);
    pixel_color[2] = sqrt(pixel_color[2]);

    frame_buffer[pixel_index] = pixel_color;
}

int main()
{
    // Image
    const auto aspect_ratio = ASPECT_RATIO;
    const int image_width = IMAGE_WIDTH;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    int num_pixels = image_width*image_height;
    size_t frame_buffer_size = num_pixels * sizeof(vec3);
    const int samples_per_pixel = SAMPLES_PER_PIXEL;
    const int max_depth = MAX_DEPTH;

    // allocate random state
    hiprandState *d_rand_state;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    dim3 blocks(image_width/TILE_SIZE+1, image_height/TILE_SIZE+1);
    dim3 threads(TILE_SIZE, TILE_SIZE);

    init_rand_state<<<blocks, threads>>>(image_width, image_height, d_rand_state);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // make our world of hittables & the camera
    hittable **d_list;
    int num_hittables = 22*22+1+3;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_list, num_hittables*sizeof(hittable *)));
    hittable **d_world;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_world, sizeof(hittable *)));
    camera **d_camera;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());


    // allocate Frame buffer
    vec3* frame_buffer;
    CUDA_SAFE_CALL(hipMallocManaged((void**)&frame_buffer, frame_buffer_size));
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // Render our buffer
    {
        Timer timer;
        render<<<blocks, threads>>>(frame_buffer, image_width, image_height, samples_per_pixel,
                                    d_camera, d_world, d_rand_state, max_depth);
        
        CUDA_SAFE_CALL(hipGetLastError());
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        std::cout << "P3\n"
                << image_width << ' ' << image_height << "\n255\n";
        for (int j = image_height-1; j >= 0; j--) {
            for (int i = 0; i < image_width; i++) {
                size_t pixel_index = j * image_width + i;
                int ir = int(255.99 * frame_buffer[pixel_index].r());
                int ig = int(255.99 * frame_buffer[pixel_index].g());
                int ib = int(255.99 * frame_buffer[pixel_index].b());
                std::cout << ir << " " << ig << " " << ib << "\n";
            }
        }
    }
    std::cerr << "\nDone.\n";
    CUDA_SAFE_CALL(hipFree(frame_buffer));
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    clean_up<<<1,1>>>(d_list,d_world, d_camera);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipFree(d_list));
    CUDA_SAFE_CALL(hipFree(d_world));
    CUDA_SAFE_CALL(hipFree(d_rand_state));
    CUDA_SAFE_CALL(hipFree(d_rand_state2));

    hipDeviceReset();
}